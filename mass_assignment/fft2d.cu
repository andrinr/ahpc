#include <blitz/array.h>
#include <fftw3.h>
#include <complex>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

using namespace blitz;
using std::complex;

extern "C++" void fft2d(Array<float, 3> grid, int nGrid, int grid_start, int grid_end, int order){
    size_t size_in_bytes = nGrid * (nGrid+2) * sizeof(float);

    int grid_size[2] = {nGrid, nGrid};
    int inembed[2] = {nGrid, nGrid};
    int onembed[2] = {nGrid, nGrid/2 + 1};
    int batch = 1;
    int odist = nGrid/2 + 1;
    int idist = 2 * odist;
    int istride = 1;
    int ostride = 1;

    hipfftHandle plan;
    hipfftPlanMany(
        &plan, 2, grid_size,
        inembed, istride, idist, onembed, ostride, odist, 
        HIPFFT_R2C, batch);

    for (int i = grid_start; i < grid_end - order; i++)
    {   
        void *device_data;

        hipMalloc(&device_data, size_in_bytes);
        hipMemcpy(device_data, &grid(i, 0, 0), size_in_bytes, hipMemcpyHostToDevice);

        hipfftExecR2C(plan, (hipfftReal*)device_data, (hipfftComplex*)device_data);

        hipMemcpy(&grid(i, 0, 0), device_data, size_in_bytes, hipMemcpyDeviceToHost);

        hipFree(device_data);
    }

    hipDeviceSynchronize();
}