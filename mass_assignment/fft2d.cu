#include <blitz/array.h>
#include <fftw3.h>
#include <complex>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>

using namespace blitz;
using std::complex;

extern "C++" void fft2d(
    Array<float, 3> grid, 
    int nGrid, 
    int grid_start, 
    int grid_end, 
    int order, 
    int n_streams) {

    int grid_size[2] = {grid.rows(), grid.cols()}; // 2D FFT of length NxN
    int inembed[2] = {grid.rows(), 2 * (grid.cols() / 2 + 1)};
    int onembed[2] = {grid.rows(), (grid.cols() / 2 + 1)};
    int batch = 1;
    int odist = grid.rows() * (grid.cols() / 2 + 1); // Output distance is in "complex"
    int idist = 2 * odist;                           // Input distance is in "real"
    int istride = 1;                                 // Elements of each FFT are adjacent
    int ostride = 1;

    size_t slab_size = sizeof(hipfftComplex) * batch * grid.rows() * (grid.cols() / 2 + 1);
    size_t work_size;

    hipfftHandle plan;
    hipfftCreate(&plan);
    hipfftSetAutoAllocation(plan, 0);
    hipfftMakePlanMany(
        plan, 2, grid_size,
        inembed, istride, idist, onembed, ostride, odist, 
        HIPFFT_R2C, batch, &work_size);

    void *d_slab_data;
    void *d_work_data;
    hipMalloc(&d_slab_data, slab_size);
    hipMalloc(&d_work_data, work_size);

    // create streams / malloc memory
    hipStream_t stream[n_streams];
    void * d_slab_data_stream[n_streams];
    void * d_work_data_stream[n_streams];

    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&stream[i]);
        hipMalloc(&d_slab_data_stream[i], slab_size);
        hipMalloc(&d_work_data_stream[i], work_size);
    }

    for (int i = grid_start; i < grid_end - order; i++)
    {   
        int stream_index = i % n_streams;

        Array<float, 2> slab = grid(i, Range::all(), Range::all());

        hipMemcpyAsync(
            d_slab_data, slab.dataFirst(), 
            slab_size, hipMemcpyHostToDevice, stream[stream_index]);

        hipfftSetStream(plan, stream[stream_index]);
        hipfftSetWorkArea(plan, d_work_data);

        hipfftExecR2C(
            plan, (hipfftReal*)d_slab_data, 
            (hipfftComplex*)d_slab_data);

        hipMemcpyAsync(
            slab.dataFirst(), d_slab_data, 
            slab_size, hipMemcpyDeviceToHost, stream[stream_index]);
    }

    hipDeviceSynchronize();

    // destroy streams / free memory
    for (int i = 0; i < n_streams; i++) {
        hipStreamDestroy(stream[i]);
        hipFree(d_slab_data_stream[i]);
        hipFree(d_work_data_stream[i]);
    }

    std::cout << "FFT 2D CUDA Done" << std::endl;

    hipfftDestroy(plan);
}