#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include <fftw3.h>
#include <complex>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

using namespace blitz;
using std::complex;

void fill_array(Array<float,2> &data) {
    // Set the grid to the sum of two sine functions
    for (int i=0; i < data.rows(); i++) {
        for (int j=0; j < data.cols(); j++) {
            float x = (float)i / 25.0; // Period of 1/4 of the box in x
            float y = (float)j / 10.0; // Period of 1/10 of the box in y
            data(i,j) = sin(2.0 * M_PI * x) + sin(2.0 * M_PI * y);
        }
    }
}

// Verify the FFT (kdata) of data by performing a reverse transform and comparing
bool validate(Array<float,2> &data,Array<std::complex<float>, 2> kdata) {
    Array<float,2> rdata(data.extent());
    fftwf_plan plan = fftwf_plan_dft_c2r_2d(data.rows(), data.cols(),
        reinterpret_cast<fftwf_complex*>(kdata.data()), rdata.data(), FFTW_ESTIMATE);
    fftwf_execute(plan);
    fftwf_destroy_plan(plan);
    rdata /= data.size(); // Normalize for the FFT
    return all(abs(data - rdata) < 1e-5);
}

int main() {
    int n = 10000;

    // Out of place
    Array<float,2> rdata1(n,n);
    Array<std::complex<float>, 2> kdata1(n, n/2 + 1);
    fftwf_plan plan1  = fftwf_plan_dft_r2c_2d(n, n,
        rdata1.data(), reinterpret_cast<fftwf_complex*>(kdata1.data()), FFTW_ESTIMATE);
    fill_array(rdata1);
    fftwf_execute(plan1);
    fftwf_destroy_plan(plan1);
    std::cout << ">>> Out of place FFT " << (validate(rdata1,kdata1)?"match":"MISMATCH") << endl;

    // in-place
    Array<float,2> raw_data2(n,n+2);
    Array<float,2> rdata2 = raw_data2(Range::all(),Range(0,n-1));
    fftwf_plan plan2  = fftwf_plan_dft_r2c_2d(n, n,
        rdata2.data(), reinterpret_cast<fftwf_complex*>(rdata2.data()), FFTW_ESTIMATE);
    fill_array(rdata2);
    fftwf_execute(plan2);
    fftwf_destroy_plan(plan2);
    Array<std::complex<float>, 2> kdata2(reinterpret_cast<std::complex<float>*>(rdata2.data()),
        shape(n, n/2 + 1),neverDeleteData);
    std::cout << ">>> In-place FFT " << (validate(rdata1,kdata2)?"match":"MISMATCH") << endl;

    // Ex 3
    Array<float,2> raw_data3(n,n);
    Array<float,2> rdata3 = raw_data3(Range::all(),Range(0,n-1));   
    fill_array(rdata3);

    Array<float,2> raw_data4(n,n);
    Array<float,2> rdata4 = raw_data4(Range::all(),Range(0,n-1));

    void *d_data3;
    size_t size_in_bytes = rdata3.size() * sizeof(float);

    hipMalloc(&d_data3, size_in_bytes);

    hipMemcpy(d_data3, rdata3.data(), size_in_bytes, hipMemcpyHostToDevice);

    hipMemcpy(rdata4.data(), d_data3, size_in_bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_data3);

    std::cout << ">>> CUDA Memory Copy " << (all(abs(rdata3 - rdata4) < 1e-5)?"match":"mismatch") << endl;

    // Ex 4
    Array<float,2> raw_data5(n,n+2);
    Array<float,2> rdata5 = raw_data5(Range::all(),Range(0,n-1));   
    fill_array(rdata5);
    Array<std::complex<float>, 2> kdata5(reinterpret_cast<std::complex<float>*>(rdata5.data()),
        shape(n, n/2 + 1),neverDeleteData);

    void *d_data5;
    size_in_bytes = n * (n+2) * sizeof(float);

    hipMalloc(&d_data5, size_in_bytes);

    hipMemcpy(d_data5, rdata5.data(), size_in_bytes, hipMemcpyHostToDevice);

    int grid_size[2] = {n, n};
    int inembed[2] = {n, n+2};
    int onembed[2] = {n, n/2 + 1};
    int batch = 1;
    int odist = n * (n/2 + 1);
    int idist = 2 * odist;
    int istride = 1;
    int ostride = 1;

    hipfftHandle plan;
    hipfftPlanMany(
        &plan, 2, grid_size,
        inembed, istride, idist, onembed, ostride, odist, 
        HIPFFT_R2C, batch);

    hipfftExecR2C(plan, (hipfftReal*)d_data5, (hipfftComplex*)d_data5);

    hipMemcpy(rdata5.data(), d_data5, size_in_bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_data5);

    std::cout << ">>> CUDA FFT " << (validate(rdata1,kdata5)?"match":"mismatch") << endl;

    return 0;
}